
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel0(float* __restrict__ X, float* __restrict__ W, float* __restrict__ T_dense) {
  float T_dense_local[96];
  __shared__ float X_shared[768];
  __shared__ float W_shared[576];
  for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 3; ++j_c_inner_init) {
      T_dense_local[(((i_c_inner_init * 3) + j_c_inner_init))] = 0.000000e+00f;
      T_dense_local[((((i_c_inner_init * 3) + j_c_inner_init) + 12))] = 0.000000e+00f;
      T_dense_local[((((i_c_inner_init * 3) + j_c_inner_init) + 24))] = 0.000000e+00f;
      T_dense_local[((((i_c_inner_init * 3) + j_c_inner_init) + 36))] = 0.000000e+00f;
      T_dense_local[((((i_c_inner_init * 3) + j_c_inner_init) + 48))] = 0.000000e+00f;
      T_dense_local[((((i_c_inner_init * 3) + j_c_inner_init) + 60))] = 0.000000e+00f;
      T_dense_local[((((i_c_inner_init * 3) + j_c_inner_init) + 72))] = 0.000000e+00f;
      T_dense_local[((((i_c_inner_init * 3) + j_c_inner_init) + 84))] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 6; ++ax0_ax1_fused_outer_outer) {
      X_shared[(((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x)))] = X[((((((((int)blockIdx.x) / 24) * 98304) + ((((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x)) / 6) * 768)) + (k_outer_outer * 6)) + (((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x)) % 6)))];
    }
    for (int ax0_ax1_fused_outer_outer1 = 0; ax0_ax1_fused_outer_outer1 < 5; ++ax0_ax1_fused_outer_outer1) {
      if (((ax0_ax1_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 576) {
        W_shared[(((ax0_ax1_fused_outer_outer1 * 128) + ((int)threadIdx.x)))] = W[((((((((int)blockIdx.x) % 24) * 73728) + ((((ax0_ax1_fused_outer_outer1 * 128) + ((int)threadIdx.x)) / 6) * 768)) + (k_outer_outer * 6)) + (((ax0_ax1_fused_outer_outer1 * 128) + ((int)threadIdx.x)) % 6)))];
      }
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 6; ++k_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 3; ++j_c_inner) {
          T_dense_local[(((i_c_inner * 3) + j_c_inner))] = (T_dense_local[(((i_c_inner * 3) + j_c_inner))] + (X_shared[(((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner))] * W_shared[(((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner))]));
          T_dense_local[((((i_c_inner * 3) + j_c_inner) + 12))] = (T_dense_local[((((i_c_inner * 3) + j_c_inner) + 12))] + (X_shared[(((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner))] * W_shared[((((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner) + 144))]));
          T_dense_local[((((i_c_inner * 3) + j_c_inner) + 24))] = (T_dense_local[((((i_c_inner * 3) + j_c_inner) + 24))] + (X_shared[(((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner))] * W_shared[((((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner) + 288))]));
          T_dense_local[((((i_c_inner * 3) + j_c_inner) + 36))] = (T_dense_local[((((i_c_inner * 3) + j_c_inner) + 36))] + (X_shared[(((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner))] * W_shared[((((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner) + 432))]));
          T_dense_local[((((i_c_inner * 3) + j_c_inner) + 48))] = (T_dense_local[((((i_c_inner * 3) + j_c_inner) + 48))] + (X_shared[((((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner) + 384))] * W_shared[(((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner))]));
          T_dense_local[((((i_c_inner * 3) + j_c_inner) + 60))] = (T_dense_local[((((i_c_inner * 3) + j_c_inner) + 60))] + (X_shared[((((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner) + 384))] * W_shared[((((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner) + 144))]));
          T_dense_local[((((i_c_inner * 3) + j_c_inner) + 72))] = (T_dense_local[((((i_c_inner * 3) + j_c_inner) + 72))] + (X_shared[((((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner) + 384))] * W_shared[((((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner) + 288))]));
          T_dense_local[((((i_c_inner * 3) + j_c_inner) + 84))] = (T_dense_local[((((i_c_inner * 3) + j_c_inner) + 84))] + (X_shared[((((((((int)threadIdx.x) >> 3) * 24) + (i_c_inner * 6)) + k_inner) + 384))] * W_shared[((((((((int)threadIdx.x) & 7) * 18) + (j_c_inner * 6)) + k_inner) + 432))]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 3; ++j_inner) {
      T_dense[((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner))] = T_dense_local[(((i_inner * 3) + j_inner))];
      T_dense[(((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner) + 24))] = T_dense_local[((((i_inner * 3) + j_inner) + 12))];
      T_dense[(((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner) + 48))] = T_dense_local[((((i_inner * 3) + j_inner) + 24))];
      T_dense[(((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner) + 72))] = T_dense_local[((((i_inner * 3) + j_inner) + 36))];
      T_dense[(((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner) + 147456))] = T_dense_local[((((i_inner * 3) + j_inner) + 48))];
      T_dense[(((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner) + 147480))] = T_dense_local[((((i_inner * 3) + j_inner) + 60))];
      T_dense[(((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner) + 147504))] = T_dense_local[((((i_inner * 3) + j_inner) + 72))];
      T_dense[(((((((((((int)blockIdx.x) / 24) * 294912) + ((((int)threadIdx.x) >> 3) * 9216)) + (i_inner * 2304)) + ((((int)blockIdx.x) % 24) * 96)) + ((((int)threadIdx.x) & 7) * 3)) + j_inner) + 147528))] = T_dense_local[((((i_inner * 3) + j_inner) + 84))];
    }
  }
}
