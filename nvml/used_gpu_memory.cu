#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <unistd.h>
#include <sys/types.h>

int main()
{
        size_t alloc_size = 30000000; 
        void * alloc_dptr = nullptr;

        nvmlDevice_t nvml_device;
        nvmlReturn_t nvml_retval;
        unsigned info_count = 0;
        std::vector < nvmlProcessInfo_t > infos;
        
        hipMalloc(&alloc_dptr, alloc_size);

        nvmlInit();
        nvmlDeviceGetHandleByIndex(0, &nvml_device);
        do {
                std::cout << "Info count: " << info_count << std::endl;
                infos = std::vector < nvmlProcessInfo_t > (info_count);
                nvml_retval = nvmlDeviceGetComputeRunningProcesses(nvml_device, &info_count, infos.data());
                std::cout << "NVML Return Val: "
                          << nvml_retval << std::endl;
                // ++info_count;
        } while (nvml_retval != NVML_SUCCESS);
        for (unsigned i = 0; i < info_count; ++i) 
        {
                std::cout << "PID " << infos[i].pid << " uses GPU memory "
                          << (infos[i].usedGpuMemory) / 1e6 << " MiB" << std::endl;
        }

        size_t free, total;
        hipMemGetInfo(&free, &total);
        std::cout << "Current PID " << getpid() << " uses GPU memory "
                  << (total - free) / 1e6 << " MiB" << std::endl;

        do {

        } while (true);

        hipFree(alloc_dptr);
        return 0;
}
