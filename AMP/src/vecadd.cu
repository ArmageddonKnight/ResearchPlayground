#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>

#include <hip/hip_fp16.h>
#include <thrust/device_vector.h>

#include <hip/hip_runtime_api.h>

template < typename DTypeT >
__device__ __forceinline__ DTypeT __2dtype(const float  val);

template < typename DTypeT, std::size_t SizeT >
__global__ void cudaVecAdd(float * const A)
{
        const unsigned g_threadIdx = threadIdx.x + blockIdx.x * blockDim.x;

        if (g_threadIdx < SizeT)
        {
                A[g_threadIdx] = __2dtype < DTypeT > (1.0f) + 
                                 __2dtype < DTypeT > (1e-4f);
        }
}

template < typename DTypeT >
std::ostream & operator<<(std::ostream & cout_,
        const thrust::device_vector < 
                DTypeT > & dev_vec)
{
        cout_ << "[" << std::endl;
        for (std::size_t i = 0; i < (dev_vec.size() + 10 - 1) / 10; ++i)
        {
                if (i * 10 >= dev_vec.size())
                {
                        break;
                }
                cout_ << "\t";
                for (std::size_t j = 0; j < 10; ++j)
                {
                        if (i * 10 + j >= dev_vec.size())
                        {
                                break;
                        }
                        cout_ << dev_vec[i * 10 + j] << ", ";
                }
                cout_ << std::endl;
        }
        cout_ << "]";
        return cout_;
}

/// \brief Top-Level Vector-Add Benchmark
template < typename DTypeT, std::size_t SizeT >
void vecadd()
{
        thrust::device_vector < float > A (SizeT);

        const std::size_t threads_per_block = 32;
        const std::size_t blocks_per_grid = (SizeT + threads_per_block - 1) / threads_per_block;
        cudaVecAdd < DTypeT, SizeT > 
                <<< blocks_per_grid, 
                    threads_per_block >>> 
                (A.data().get());
        checkCudaErrors(hipDeviceSynchronize());
        std::cout << A << std::endl;
}

int main()
{
        vecadd < float, 100 > ();
        vecadd < half,  100 > ();

        return 0;
}

template <>
float __2dtype < float > (const float val) { return val; }
template <>
half  __2dtype < half >  (const float val) { return __float2half(val); }
