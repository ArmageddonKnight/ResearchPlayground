#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>

#include <hip/hip_fp16.h>
#include <thrust/device_vector.h>

#include <hip/hip_runtime_api.h>

template < typename DTypeT, std::size_t N >
__global__ void cudaVecAdd(
        const DTypeT * const A,
        const DTypeT * const B,
              DTypeT * const C)
{
        const unsigned g_threadIdx = threadIdx.x + blockIdx.x * blockDim.x;

        if (g_threadIdx < N)
        {
                C[g_threadIdx] = A[g_threadIdx] + B[g_threadIdx];
        }
}

template < typename DTypeT >
std::ostream & operator<<(std::ostream & cout_,
        const thrust::device_vector < 
                DTypeT > & dev_vec)
{
        cout_ << "[" << std::endl;
        for (std::size_t i = 0; i < (dev_vec.size() + 10 - 1) / 10; ++i)
        {
                if (i * 10 >= dev_vec.size())
                {
                        break;
                }
                cout_ << "\t";
                for (std::size_t j = 0; j < 10; ++j)
                {
                        if (i * 10 + j >= dev_vec.size())
                        {
                                break;
                        }
                        cout_ << (dev_vec[i * 10 + j]) << ", ";
                }
                cout_ << std::endl;
        }
        cout_ << "]";
        return cout_;
}

/// \brief Top-Level Vector-Add Benchmark
template < typename DTypeT, std::size_t SizeT >
void vecadd()
{
        thrust::device_vector < DTypeT > A (SizeT, 1);
        thrust::device_vector < DTypeT > B (SizeT, 1);
        thrust::device_vector < DTypeT > C (SizeT, 0);

        const std::size_t threads_per_block = 32;
        const std::size_t blocks_per_grid = (SizeT + threads_per_block - 1) / threads_per_block;
        cudaVecAdd < DTypeT, SizeT > 
                <<< blocks_per_grid, 
                    threads_per_block >>> 
                (A.data().get(), 
                 B.data().get(), 
                 C.data().get());
        checkCudaErrors(hipDeviceSynchronize());
        std::cout << C << std::endl;
}

int main()
{
        vecadd < float, 100 > ();
        vecadd < half,  100 > ();

        return 0;
}
