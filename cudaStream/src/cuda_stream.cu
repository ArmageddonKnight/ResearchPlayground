#include <iostream>
// Program Options
#include <boost/program_options.hpp>
// CUDA Headers
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// Helper Functions
#include "hip/hip_runtime_api.h"

int main(int argc, char * argv[])
{
        using namespace boost::program_options;

        options_description desc ("cudaStream Microbenchmark Program Options");

        desc.add_options()
                ("help,h", "Helping Message")
                ("batch-size,b", value < std::size_t > (), "Batch Size")
                ("input-size,i", value < std::size_t > (), "Input Size");
        variables_map vm;
        store(parse_command_line(argc, argv, desc), vm);

        if (!vm.count("batch-size") || !vm.count("input-size"))
        {
                std::cerr << "[ERROR] Tensor shapes must be provided." << std::endl;
                exit(EXIT_FAILURE);
        } 

        std::size_t batch_size = vm["batch-size"].as < std::size_t > (), 
                    input_size = vm["input-size"].as < std::size_t > ();
        std::size_t state_size = 4 * input_size;

        std::cout << "[INFO] B = " << batch_size << ", "
                            "I = " << input_size << ", "
                            "H = " << state_size << std::endl;

        thrust::device_vector < float > I     (batch_size * input_size, 1.0f);
        thrust::device_vector < float > H     (batch_size * input_size, 1.0f);
        thrust::device_vector < float > W_I2H (state_size * input_size, 1.0f);
        thrust::device_vector < float > W_H2H (state_size * input_size, 1.0f);
        thrust::device_vector < float > Y_I   (batch_size * state_size, 0.0f);
        thrust::device_vector < float > Y_H   (batch_size * state_size, 0.0f);

        // =====================================================================
        // Baseline (Default Stream)
        // =====================================================================

        struct cublasHandleWrapper
        {
        private:
                hipblasHandle_t handle;
        public:
                cublasHandleWrapper() { checkCudaErrors(hipblasCreate(&handle)); }
               ~cublasHandleWrapper() { checkCudaErrors(hipblasDestroy(handle)); }
                operator hipblasHandle_t() { return handle; }
        };

        cublasHandleWrapper cublas_handle; const float alpha = 1.0f, beta = 0.0f;
        
        // =====================================================================

        checkCudaErrors(hipblasSetStream(cublas_handle, nullptr));
        checkCudaErrors(hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_T, 
                        HIPBLAS_OP_N,
                        state_size, 
                        batch_size,
                        input_size,
                        &alpha, 
                        W_I2H.data().get(), input_size, 
                        I    .data().get(), input_size,
                        &beta, 
                        Y_I  .data().get(), state_size));
        checkCudaErrors(hipblasSetStream(cublas_handle, nullptr));
        checkCudaErrors(hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_T, 
                        HIPBLAS_OP_N,
                        state_size, 
                        batch_size,
                        input_size,
                        &alpha, 
                        W_H2H.data().get(), input_size, 
                        H    .data().get(), input_size,
                        &beta, 
                        Y_H  .data().get(), state_size));
        checkCudaErrors(hipStreamSynchronize(nullptr));

        // =====================================================================
        // Multi-Stream Version
        // =====================================================================

        struct cudaStreamWrapper
        {
                hipStream_t stream;
                
                cudaStreamWrapper(int priority = 0)
                {
                        checkCudaErrors(hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, priority));
                }
               ~cudaStreamWrapper() { checkCudaErrors(hipStreamDestroy(stream)); }
                operator hipStream_t() { return stream; }
        };

        cudaStreamWrapper i2h_stream, h2h_stream;

        // =====================================================================

        checkCudaErrors(hipblasSetStream(cublas_handle, i2h_stream));
        checkCudaErrors(hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_T, 
                        HIPBLAS_OP_N,
                        state_size, 
                        batch_size,
                        input_size,
                        &alpha, 
                        W_I2H.data().get(), input_size, 
                        I    .data().get(), input_size,
                        &beta, 
                        Y_I  .data().get(), state_size));
        checkCudaErrors(hipblasSetStream(cublas_handle, h2h_stream));
        checkCudaErrors(hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_T, 
                        HIPBLAS_OP_N,
                        state_size, 
                        batch_size,
                        input_size,
                        &alpha, 
                        W_H2H.data().get(), input_size, 
                        H    .data().get(), input_size,
                        &beta, 
                        Y_H  .data().get(), state_size));

        checkCudaErrors(hipStreamSynchronize(i2h_stream));
        checkCudaErrors(hipStreamSynchronize(h2h_stream));

        return 0;
}
