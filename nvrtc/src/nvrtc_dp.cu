#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>

#include <hip/hip_runtime.h>
#include <nvrtc_helper.h>  // hip/hiprtc.h

const char * const nvrtc_dp = "                                                 \n\
__global__ void child(float * out, std::size_t n)                               \n\
{                                                                               \n\
        std::size_t g_threadIdx = blockIdx.x +                                  \n\
                                  blockDim.x * threadIdx.x;                     \n\
        if (g_threadIdx < n)                                                    \n\
        {                                                                       \n\
                out[g_threadIdx] = g_threadIdx;                                 \n\
        }                                                                       \n\
}                                                                               \n\
__global__ void parent(float * out,                                             \n\
        std::size_t n,                                                          \n\
        std::size_t num_blocks,                                                 \n\
        std::size_t num_threads)                                                \n\
{                                                                               \n\
        child <<< num_blocks, num_threads >>> (out, n);                         \n\
        hipDeviceSynchronize();                                                \n\
}                                                                               \n";

int main(int argc, char * argv[])
{
        std::size_t num_blocks = 32, num_threads = 128;

        hiprtcProgram prog;
}
